#include "hip/hip_runtime.h"
#include <cube.h>
#include <cube_matrix.h>
#include <cube_matlab.h>
#include <cube_blas.h>
#include <cube_math.h>
#include <cube_ica_kernels.h>

#include <stdio.h>
#include <string.h>

#include "cube_private.h"


#define min(a,b) ((a)>(b)?(b):(a))

void
matrix_dump (double *matrix, int m, int n, int m_max, int n_max, int m_s = 0, int n_s = 0)
{
  int row, col;
  int im, in;

  if (matrix == NULL)
    return;

  im = min (m, m_max + m_s);
  in = min (n, n_max + n_s);

  for (row = m_s; row < im; row++)
    {
      printf ("[");
      for (col = n_s; col < in; col++)
	{
	  int pos = (col * m) + row;
	  printf (" %0.1lf", matrix[pos]); //A[IDX2F(row, col, n)]);
	}

      printf ("], \n");
    }
  printf ("\n");
}

int
main (int argc, char **argv)
{
  hipError_t res;
  cube_matrix_t *diag;
  double *x, *X, *iter;
  cube_t *ctx;
  int n;

  ctx = cube_context_new (0);
  
  n = 50;

  diag = cube_matrix_new_on_device (ctx, n, n);
  x = (double *) cube_malloc_device (ctx, n * sizeof (double));

  for (iter = x; iter < (x + n); iter++)
    {
      double d = 0.5;
      cube_memcpy (ctx, iter, &d, sizeof (double), CMK_HOST_2_DEVICE);
    }

  cube_matrix_diag (ctx, diag, 1, 0.25, x, 1);

  res = hipPeekAtLastError ();
  cube_cuda_check (ctx, res);

  X = (double *) malloc (n * n * sizeof (double));
  cube_memcpy (ctx, X, diag->dev_ptr, n * n * sizeof (double), CMK_DEVICE_2_HOST);

  matrix_dump (X, n, n, 50, 50);

  free (X);
  cube_free_device (ctx, x);
  cube_matrix_destroy (ctx, diag);
  cube_context_destroy (&ctx);

  return 0;
}
