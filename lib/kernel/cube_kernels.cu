#include "hip/hip_runtime.h"
#include "cube.h"
#include "cube_blas.h"
#include "cube_matrix.h"

#include "cube_kernels.h"
#include "cube_private.h"

#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ double
d_inv (double x, int inv)
{
  return inv ? 1.0 / x : x;
}

__global__ void
k_diag (int n, double *D, int ldd, int inv, double alpha, double *x, int incx)
{
  extern __shared__ double smem[];
  double *s;

  int     global_x, global_y, lid, gid;

  /* calculate global and local ids */
  global_x = (blockDim.x * blockIdx.x) + threadIdx.y; //n
  global_y = (blockDim.y * blockIdx.y) + threadIdx.x; //m

  gid = (ldd * global_x) + global_y;
  lid = (threadIdx.y * blockDim.x) + threadIdx.x;

  smem[lid] = 0;

  if (blockIdx.x == blockIdx.y && threadIdx.x < warpSize && global_y < n)
    {
      s = &smem[blockDim.x * blockDim.y];
      s[threadIdx.x] = x[(blockIdx.x * blockDim.x + threadIdx.x) * incx];
      smem[threadIdx.x * blockDim.x + threadIdx.x] = d_inv (s[threadIdx.x] * alpha, inv);
    }

  if (global_x < n && global_y < n)
    D[gid] = smem[lid];
}

void
cube_gpu_diag (cube_t *ctx, int n, double *diag, int ldd, int inv, double alpha, double *x, int incx)
{
  size_t  smem;
  dim3    block, grid;

  if (! cube_context_check (ctx))
    return;

  block.x = 32;
  block.y = 32;

  grid.x = ceil (n / (double) block.x);
  grid.y = grid.x;

  smem = (block.x + 1) * block.y * sizeof (double);

  k_diag<<<block, grid, smem>>> (n, diag, ldd, inv, alpha, x, incx);
}

